#include "hip/hip_runtime.h"
#include "yolov3layer.h"



__device__ float Logist(float data)
{
    return 1.0f / (1.0f + expf(-data));
}

__global__void CalDetection(const float *input,
                            float *output,
                            int noElements,
                            int yoloWidth,
                            int yoloHeight,
                            const float anchors[3*2],
                            int classes, int outputElem)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx >= noElements)
        return;

    int total_grid = yoloWidth * yoloHeight;
    int bnIdx = idx / total_grid;
    idx = idx - total_grid * bnIdx;
    int info_len_i = 5 + classes;
    const float* curInput = input + bnIdx * (info_len_i * total_grid * CHECK_COUNT);

    for (int k = 0; k < 3; ++k)
    {
        int class_id = 0;
        float max_cls_prob = 0.0;
        for (int i = 5; i < info_len_i; ++i)
        {
            float p = Logist(curInput[idx + k * info_len_i * total_grid + i * total_grid]);
            if (p > max_cls_prob)
            {
                max_cls_prob = p;
                class_id = i - 5;
            }
        }
        float box_prob = Logist(curInput[idx + k * info_len_i * total_grid + 4 * total_grid]);
        if (max_cls_prob < IGNORE_THRESH || box_prob < IGNORE_THRESH)
            continue;

        float *res_count = output + bnIdx*outputElem;
        int count = (int)atomicAdd(res_count, 1);

        if (count >= MAX_OUTPUT_BBOX_COUNT)
            return;

        char* data = (char * )res_count + sizeof(float) + count*sizeof(Detection);
        Detection* det =  (Detection*)(data);

        int row = idx / yoloWidth;
        int col = idx % yoloWidth;

        //Location
        det->bbox[0] = (col + Logist(curInput[idx + k * info_len_i * total_grid + 0 * total_grid])) * INPUT_W / yoloWidth;
        det->bbox[1] = (row + Logist(curInput[idx + k * info_len_i * total_grid + 1 * total_grid])) * INPUT_H / yoloHeight;
        det->bbox[2] = expf(curInput[idx + k * info_len_i * total_grid + 2 * total_grid]) * anchors[2*k];
        det->bbox[3] = expf(curInput[idx + k * info_len_i * total_grid + 3 * total_grid]) * anchors[2*k + 1];
        det->det_confidence = box_prob;
        det->class_id = class_id;
        det->class_confidence = max_cls_prob;
    }

}